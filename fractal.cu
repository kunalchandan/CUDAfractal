
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <complex>
#include <hip/hip_complex.h>

#include <X11/Xlib.h>
#include <X11/Xos.h>
#include <X11/Xutil.h>

#include <cstdint>
#include <iomanip>
#include <iostream>
#include <tuple>
#include <vector>

const uint32_t WIDTH = 2048, HEIGHT = 2048;
const uint32_t ITER_MAX = 512;

void mandelbrot_single_thread(int n, int* mbr) {
    for(int i = 0; i < n; i++) {
        // Index from the top left, using row-major order/x-major order
        uint32_t x_index = (i % WIDTH);
        uint32_t y_index = (i / WIDTH);

        float calculated_position_x = (x_index / (1.0 * WIDTH)) - 0.5;
        float calculated_position_y = (y_index / (1.0 * WIDTH)) - 0.5;

        auto z_0 = std::complex(calculated_position_x, calculated_position_y);
        auto z_n = std::complex(calculated_position_x, calculated_position_y);

        int m_i = 0;
        for(m_i = 0; m_i < ITER_MAX; m_i++) {
            z_n = std::pow(z_n, std::complex(2.0f, 0.0f)) + z_0;
            if(std::norm(z_n) > 2) {
                break;
            }
        }
        mbr[i] = m_i;
    }
}

__global__ void mandelbrot_single_thread_gpu(int n, int* mbr) {
    for(int i = 0; i < n; i++) {
        // Index from the top left, using row-major order/x-major order
        uint32_t x_index = (i % WIDTH);
        uint32_t y_index = (i / WIDTH);

        float calculated_position_x = (x_index / (1.0 * WIDTH)) - 0.5;
        float calculated_position_y = (y_index / (1.0 * HEIGHT)) - 0.5;

        hipFloatComplex z_0 = make_hipFloatComplex(calculated_position_x, calculated_position_y);
        hipFloatComplex z_n = make_hipFloatComplex(calculated_position_x, calculated_position_y);

        int m_i = 0;
        for(m_i = 0; m_i < ITER_MAX; m_i++) {
            z_n = hipCaddf(hipCmulf(z_n, z_n), z_0);
            if(hipCabsf(z_n) > 2.0) {
                break;
            }
        }
        mbr[i] = m_i;
    }
}

__global__ void
mandelbrot_multi_thread_gpu(int n, int* mbr, uint16_t thread_blocks, uint16_t threads) {
    int tid          = blockIdx.x * blockDim.x + threadIdx.x;
    int section_size = (n) / (thread_blocks * threads);
    for(int i = tid * section_size; i < (tid + 1) * section_size; i++) {
        // Index from the top left, using row-major order/x-major order
        uint32_t x_index = (i % WIDTH);
        uint32_t y_index = (i / WIDTH);

        float calculated_position_x = (x_index / (1.0 * WIDTH)) - 0.5;
        float calculated_position_y = (y_index / (1.0 * HEIGHT)) - 0.5;

        hipFloatComplex z_0 = make_hipFloatComplex(calculated_position_x, calculated_position_y);
        hipFloatComplex z_n = make_hipFloatComplex(calculated_position_x, calculated_position_y);

        int m_i = 0;
        for(m_i = 0; m_i < ITER_MAX; m_i++) {
            z_n = hipCaddf(hipCmulf(z_n, z_n), z_0);
            if(hipCabsf(z_n) > 2.0) {
                break;
            }
        }
        mbr[i] = m_i;
    }
}

void drawCalculation(Display* di, Window wi, GC gc, int* mbr, std::vector<uint32_t> color_lookup) {
    for(int x = 0; x < WIDTH; x++) {
        for(int y = 0; y < HEIGHT; y++) {
            XSetForeground(di, gc, color_lookup[mbr[(y * WIDTH) + x]]);
            XDrawPoint(di, wi, gc, x, y);
        }
    }
}

std::vector<uint32_t> generate_color_lookup() {
    std::vector<uint32_t> table = std::vector<uint32_t>(ITER_MAX);
    for(int x = 0; x < ITER_MAX; x++) {
        // uint32_t l2 = (std::log2(x));
        uint32_t l2 = (x * 255.0) / ITER_MAX;
        // std::printf("%i", l2);
        table[x] = (l2) | (l2 << 8) | (l2 << 16);
    }
    return table;
}

std::tuple<Display*, Window, GC> OpenDisplay() {
    //Open Display
    Display* di = XOpenDisplay(getenv("DISPLAY"));
    if(di == NULL) {
        printf("Couldn't open display.\n");
        exit(-1);
    }

    //Create Window
    int const x = 0, y = 0, border_width = 1;
    int       sc = DefaultScreen(di);
    Window    ro = DefaultRootWindow(di);
    Window    wi = XCreateSimpleWindow(
        di, ro, x, y, WIDTH, HEIGHT, border_width, BlackPixel(di, sc), WhitePixel(di, sc));
    XMapWindow(di, wi); //Make window visible
    XStoreName(di, wi, "Mandelbrot Fractal"); // Set window title

    //Prepare the window for drawing
    GC gc = XCreateGC(di, ro, 0, NULL);

    return std::make_tuple(di, wi, gc);
}

void userInput(Display* di, Window wi, GC gc, int* mbr, std::vector<uint32_t> color_lookup) {
    //Select what events the window will listen to
    XSelectInput(di, wi, KeyPressMask | ExposureMask);
    XEvent ev;
    int    quit = 0;
    while(!quit) {
        int a = XNextEvent(di, &ev);
    }
    XFreeGC(di, gc);
    XDestroyWindow(di, wi);
    XCloseDisplay(di);
}

int main(void) {
    auto display_open_result = OpenDisplay();
    auto di                  = std::get<0>(display_open_result);
    auto wi                  = std::get<1>(display_open_result);
    auto gc                  = std::get<2>(display_open_result);

    int N = WIDTH * HEIGHT;

    int* mbr;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&mbr, N * sizeof(float));
    // mandelbrot_single_thread_gpu<<<1, 1>>>(N, mbr);
    uint16_t thread_blocks = 4;
    uint16_t threads       = 256;

    mandelbrot_multi_thread_gpu<<<thread_blocks, threads>>>(N, mbr, thread_blocks, threads);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Display calculated mandelbrot set:
    std::vector<uint32_t> color_lookup_table = generate_color_lookup();

    drawCalculation(di, wi, gc, mbr, color_lookup_table);

    userInput(di, wi, gc, mbr, color_lookup_table);
    // Free memory
    hipFree(mbr);
}